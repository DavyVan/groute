#include "hip/hip_runtime.h"
// Groute: An Asynchronous Multi-GPU Programming Framework
// http://www.github.com/groute/groute
// Copyright (c) 2017, A. Barak
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice,
//   this list of conditions and the following disclaimer.
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
// * Neither the names of the copyright holders nor the names of its 
//   contributors may be used to endorse or promote products derived from this
//   software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
#include <vector>
#include <algorithm>
#include <thread>
#include <memory>
#include <random>

#include <gflags/gflags.h>

#include <groute/device/cta_scheduler.cuh>
#include <groute/graphs/csr_graph.h>
#include <groute/dwl/distributed_worklist.cuh>
#include <groute/dwl/workers.cuh>

#include <utils/graphs/traversal.h>

#include "pr_common.h"



namespace pr {

    struct RankData
    {
        index_t node;
        rank_t rank;

        __host__ __device__ __forceinline__ RankData(index_t node, rank_t rank) : node(node), rank(rank) { }
        __host__ __device__ __forceinline__ RankData() : node(UINT_MAX), rank(-1.0f) { }
    };

    typedef index_t local_work_t;
    typedef RankData remote_work_t;

    template<typename TGraph, typename ResidualDatum>
    __global__ void PageRankInitKernel(TGraph graph, ResidualDatum residual)
    {
        unsigned tid = TID_1D;
        unsigned nthreads = TOTAL_THREADS_1D;

        index_t start_node = graph.owned_start_node();
        index_t end_node = start_node + graph.owned_nnodes();

        for (index_t node = start_node + tid; node < end_node; node += nthreads)
        {
            residual[node] = 1.0 - ALPHA;
        }
    }

    /*
    ---- PageRank Algorithm (data-driven, push-based) ----

    Based on Algorithm 4. in http://www.cs.utexas.edu/~inderjit/public_papers/scalable_pagerank_europar15.pdf
    Terminology: T[v]: All outgoing neighbors of node v
                 S[v]: All incoming neighbors of node v

    Input: graph G = (V, E), alpha, epsilon
    Output: PageRank x
    
    Initialize x : x[i] = 1 - alpha                         // Ranks vector
    Initialize r : r[i] = 0                                 // Residual vector

    // Initialization round
    for v in V:
        for w in T[v]:                                      // We perform also the initialization round with push-style, unlike the paper 
            r[w] = r[w] + ( (1-alpha)*alpha / |T[v]| )      // This line fuses lines 5 and 7 in the paper (maybe with some numeric cost)
        worklist.push(v)

    // Relaxation
    while worklist.not_empty
        v = worklist.pop()
        x[v] = x[v] + r[v]
        for w in T[v]:  
            o = r[w]
            r[w] = r[w] + r[v]*alpha/|T[v]|
            if r[w] >= epsilon and o < epsilon
                worklist.push(w)
        r[v] = 0

    // L1 Normalization
    x = x / ||x||_1
    */

    //template<
    //    typename TGraph, typename RankDatum, typename ResidualDatum,  typename WorkTarget>
    //__global__ void PageRankInitKernel(
    //    TGraph graph, RankDatum current_ranks, ResidualDatum residual, WorkTarget remote_work_target)
    //{
    //    unsigned tid = TID_1D;
    //    unsigned nthreads = TOTAL_THREADS_1D;

    //    index_t start_node = graph.owned_start_node();
    //    index_t end_node = start_node + graph.owned_nnodes();

    //    for (index_t node = start_node + tid; node < end_node; node += nthreads)
    //    {
    //        current_ranks[node] = 1.0 - ALPHA;

    //        index_t
    //            begin_edge = graph.begin_edge(node),
    //            end_edge = graph.end_edge(node),
    //            out_degree = end_edge - begin_edge;

    //        if (out_degree == 0) continue;

    //        rank_t update = ((1.0 - ALPHA) * ALPHA) / out_degree;

    //        for (index_t edge = begin_edge; edge < end_edge; ++edge)
    //        {
    //            index_t dest = graph.edge_dest(edge);
    //            if( atomicAdd(residual.get_item_ptr(dest), update) == 0 )
    //            {
    //                if (!graph.owns(dest)) // Push only remote nodes into the worklist since we process all 
    //                {
    //                    remote_work_target.append_work(dest);
    //                }
    //            }
    //        }
    //    }
    //}

    template<bool CTAScheduling = true> 
    /// PR work with Collective Thread Array scheduling for exploiting nested parallelism 
    struct PageRankWork
    {
        template<
            typename WorkSource, typename WorkTarget, 
            typename TGraph, typename ResidualDatum, typename RankDatum>
        __device__ static void work(
            const WorkSource& work_source, WorkTarget& work_target,
            const TGraph& graph, ResidualDatum& residual, RankDatum& current_ranks
            )
        {
            uint32_t tid = TID_1D;
            uint32_t nthreads = TOTAL_THREADS_1D;

            uint32_t work_size = work_source.get_size();
            uint32_t work_size_rup = round_up(work_size, blockDim.x) * blockDim.x; // we want all threads in active blocks to enter the loop

            for (uint32_t i = 0 + tid; i < work_size_rup; i += nthreads)
            {
                groute::dev::np_local<rank_t> np_local = { 0, 0, 0.0 };

                if (i < work_size)
                {
                    index_t node = work_source.get_work(i);
                    rank_t res = atomicExch(residual.get_item_ptr(node), 0);

                    if (res > 0)
                    {
                        current_ranks[node] += res;

                        np_local.start = graph.begin_edge(node);
                        np_local.size = graph.end_edge(node) - np_local.start;

                        if (np_local.size > 0) // Skip zero-degree nodes 
                        {
                            rank_t update = res * ALPHA / np_local.size;
                            np_local.meta_data = update;
                        }
                    }
                }

                groute::dev::CTAWorkScheduler<rank_t>::template schedule(
                    np_local,
                    [&work_target, &graph, &residual](index_t edge, rank_t update)
                    {
                        index_t dest = graph.edge_dest(edge);
                        rank_t prev = atomicAdd(residual.get_item_ptr(dest), update);

                        // The EPSILON test must be decided by the owner, so if 
                        // dest belongs to another device the threshold is 0
                        rank_t threshold = graph.owns(dest) ? EPSILON : 0;

                        if (prev <= threshold && prev + update > threshold)
                        {
                            work_target.append_work(dest);
                        }
                    }
                );
            }
        }
    };
       
    template<>
    /// PR work without CTA support
    struct PageRankWork< false >
    {
        template<
            typename WorkSource, typename WorkTarget, 
            typename TGraph, typename ResidualDatum, typename RankDatum>
        __device__ static void work(
            const WorkSource& work_source, WorkTarget& work_target,
            const TGraph& graph, ResidualDatum& residual, RankDatum& current_ranks
            )
        {
            uint32_t tid = TID_1D;
            uint32_t nthreads = TOTAL_THREADS_1D;

            uint32_t work_size = work_source.get_size();

            for (uint32_t i = 0 + tid; i < work_size; i += nthreads)
            {
                index_t node = work_source.get_work(i);

                rank_t res = atomicExch(residual.get_item_ptr(node), 0);
                if (res == 0) continue; // might happen if work_source has duplicates  

                current_ranks[node] += res;

                index_t
                    begin_edge = graph.begin_edge(node),
                    end_edge = graph.end_edge(node),
                    out_degree = end_edge - begin_edge;

                if (out_degree == 0) continue;

                rank_t update = res * ALPHA / out_degree;

                for (index_t edge = begin_edge; edge < end_edge; ++edge)
                {
                    index_t dest = graph.edge_dest(edge);
                    rank_t prev = atomicAdd(residual.get_item_ptr(dest), update);

                    // The EPSILON test must be decided by the owner, so if 
                    // dest belongs to another device the threshold is 0
                    rank_t threshold = graph.owns(dest) ? EPSILON : 0;

                    if (prev <= threshold && prev + update > threshold)
                    {
                        work_target.append_work(dest);
                    }
                }
            }
        }
    };

    /*
    template<
        typename TGraph,
        typename RankDatum,
        typename ResidualDatum>
        __global__ void PageRankFusedInit(TGraph graph,
        RankDatum current_ranks, ResidualDatum residual,
        groute::dev::CircularWorklist<local_work_t> rwl_in,   // prepending work here
        groute::dev::CircularWorklist<remote_work_t> rwl_out,  // appending work here
        volatile int*     host_high_work_counter,
        volatile int*     host_low_work_counter,
        volatile int *    send_signal_ptr,
        hipcub::GridBarrier gbar)
    {
        unsigned tid = TID_1D;
        unsigned nthreads = TOTAL_THREADS_1D;

        index_t start_node = graph.owned_start_node();
        index_t end_node = start_node + graph.owned_nnodes();

        // Do init step 1
        //
        for (index_t node = start_node + tid; node < end_node; node += nthreads)
        {
            current_ranks[node] = 1.0 - ALPHA;

            index_t
                begin_edge = graph.begin_edge(node),
                end_edge = graph.end_edge(node),
                out_degree = end_edge - begin_edge;

            if (out_degree == 0) continue;

            rank_t update = ((1.0 - ALPHA) * ALPHA) / out_degree;

            for (index_t edge = begin_edge; edge < end_edge; ++edge)
            {
                index_t dest = graph.edge_dest(edge);

                if (graph.owns(dest))
                {
                    atomicAdd(residual.get_item_ptr(dest), update);
                }
                else // we only append remote nodes, since all owned nodes are processed at step 2
                {
                    // Write directly to remote out without atomics
                    rwl_out.append_warp(RankData(dest, update));
                }
            }
        }

        gbar.Sync();

        int prev_start;

        // Transmit work
        if (GTID == 0)
        {
            uint32_t remote_work_count = rwl_out.get_alloc_count_and_sync();
            if (remote_work_count > 0) groute::dev::Signal::Increase(send_signal_ptr, remote_work_count);

            prev_start = rwl_in.get_start();
        }

        gbar.Sync();

        // Do init step 2
        //
        PageRankWork<TGraph, RankDatum, ResidualDatum>::work(
            groute::dev::WorkSourceRange<index_t>(
            graph.owned_start_node(),
            graph.owned_nnodes()),
            rwl_in, rwl_out,
            graph, current_ranks, residual
            );

        gbar.Sync();

        // Transmit and report work
        if (GTID == 0)
        {
            uint32_t remote_work_count = rwl_out.get_alloc_count_and_sync();
            if (remote_work_count > 0) groute::dev::Signal::Increase(send_signal_ptr, remote_work_count);

            __threadfence();
            // Report work
            *host_high_work_counter = rwl_in.get_start_diff(prev_start) - graph.owned_nnodes();
            *host_low_work_counter = 0;
        }
    }
    */

    struct DWCallbacks
    {
    private:
        groute::graphs::dev::CSRGraphSeg m_graph_seg;
        groute::graphs::dev::GraphDatum<rank_t> m_residual;

    public:
        template<typename...UnusedData>
        DWCallbacks(
            const groute::graphs::dev::CSRGraphSeg& graph_seg,
            const groute::graphs::dev::GraphDatum<rank_t>& residual,
            const groute::graphs::dev::GraphDatumSeg<rank_t>& current_ranks,
            UnusedData&... data)
            :
            m_graph_seg(graph_seg),
            m_residual(residual)
        {
        }

        DWCallbacks(
            const groute::graphs::dev::CSRGraphSeg& graph_seg,
            const groute::graphs::dev::GraphDatum<rank_t>& residual)
            :
            m_graph_seg(graph_seg),
            m_residual(residual)
        {
        }

        DWCallbacks() { }

        __device__ __forceinline__ groute::SplitFlags on_receive(const remote_work_t& work)
        {
            if (m_graph_seg.owns(work.node))
            {
                rank_t prev = atomicAdd(m_residual.get_item_ptr(work.node), work.rank);
                return (prev + work.rank > EPSILON && prev <= EPSILON)
                    ? groute::SF_Take
                    : groute::SF_None;
            }

            return groute::SF_Pass;
        }

        __device__ __forceinline__ bool should_defer(const local_work_t& work, const rank_t& global_threshold)
        {
            return false; // TODO: How can soft-priority be helpfull for PR?
        }

        __device__ __forceinline__ groute::SplitFlags on_send(local_work_t work)
        {
            return (m_graph_seg.owns(work))
                ? groute::SF_Take
                : groute::SF_Pass;
        }

        __device__ __forceinline__ remote_work_t pack(local_work_t work)
        {
            return RankData(work, atomicExch(m_residual.get_item_ptr(work), 0));
        }

        __device__ __forceinline__ local_work_t unpack(const remote_work_t& work)
        {
            return work.node;
        }
    };

    struct Algo
    {
        static const char* NameLower()      { return "pr"; }
        static const char* Name()           { return "PR"; }

        static void HostInit(
            utils::traversal::Context<pr::Algo>& context,
            groute::graphs::multi::CSRGraphAllocator& graph_manager,
            groute::IDistributedWorklist<local_work_t, remote_work_t>& distributed_worklist)
        {
            // PR starts with all nodes
            distributed_worklist.ReportInitialWork(context.host_graph.nnodes, groute::Endpoint::HostEndpoint(0)); 
        }

        template<typename TGraph, typename ResidualDatum, typename RankDatum, typename...UnusedData>
        static void DeviceMemset(groute::Stream& stream, TGraph graph, ResidualDatum residual, RankDatum ranks)
        {
            GROUTE_CUDA_CHECK(
                hipMemsetAsync(residual.data_ptr, 0, residual.size * sizeof(rank_t), stream.hip_stream));
            GROUTE_CUDA_CHECK(
                hipMemsetAsync(ranks.data_ptr, 0, ranks.size * sizeof(rank_t), stream.hip_stream));
        }

        template<typename TGraph, typename ResidualDatum, typename RankDatum, typename...UnusedData>
        static void DeviceInit(
            groute::Endpoint endpoint, groute::Stream& stream, 
            groute::IDistributedWorklist<local_work_t, remote_work_t>& distributed_worklist, 
            groute::IDistributedWorklistPeer<local_work_t, remote_work_t, DWCallbacks>* peer, 
            TGraph graph, ResidualDatum residual, RankDatum ranks)
        {   
            auto& workspace = peer->GetLocalWorkspace(0);
            DWCallbacks callbacks = peer->GetDeviceCallbacks();

            dim3 grid_dims, block_dims;

            // Init residual values
            KernelSizing(grid_dims, block_dims, graph.owned_nnodes());
            PageRankInitKernel <<< grid_dims, block_dims, 0, stream.hip_stream >>>(graph, residual);

            KernelSizing(grid_dims, block_dims, graph.owned_nnodes());
            groute::WorkKernel <groute::dev::WorkSourceRange<index_t>, local_work_t, remote_work_t, DWCallbacks, PageRankWork<>, TGraph, ResidualDatum, RankDatum>

                <<< grid_dims, block_dims, 0, stream.hip_stream >>> (

                    groute::dev::WorkSourceRange<index_t>(graph.owned_start_node(), graph.owned_nnodes()),
                    workspace.DeviceObject(),
                    callbacks,
                    graph, residual, ranks
                    );

            auto output_seg = workspace.ToSeg(stream);
            int work = output_seg.GetSegmentSize(); 
            peer->SplitSend(output_seg, stream); 

            workspace.ResetAsync(stream); 
            distributed_worklist.ReportWork(work, graph.owned_nnodes(), endpoint);

            ///

            //auto& input_worklist = peer->GetLocalInputWorklist();
            //auto& temp_worklist = peer->GetLocalWorkspace(0); // local output worklist
            //
            //m_problem.Init__Multi__(temp_worklist, stream);
            //
            //auto seg1 = temp_worklist.ToSeg(stream);

            //// report work
            //distributed_worklist.ReportWork((int)seg1.GetSegmentSize(), (int)m_problem.m_graph.owned_nnodes(), endpoint );

            //peer->SplitSend(seg1, stream); // call split-send
            //
            //temp_worklist.ResetAsync(stream.hip_stream); // reset the temp output worklist

            //// First relax is a special case, starts from all owned nodes
            //m_problem.Relax__Multi__(
            //    groute::dev::WorkSourceRange<index_t>(
            //        m_problem.m_graph.owned_start_node(),
            //        m_problem.m_graph.owned_nnodes()),
            //        temp_worklist, stream);

            //auto seg2 = temp_worklist.ToSeg(stream);
            //
            //// report work
            //distributed_worklist.ReportWork((int)seg2.GetSegmentSize(), 0, endpoint);

            //peer->SplitSend(seg2, stream); // call split-send
            //temp_worklist.ResetAsync(stream.hip_stream); // reset the temp output worklist
        }

        template<
            typename TGraphAllocator, typename ResidualDatum, typename RankDatum, typename...UnusedData>
        static const std::vector<rank_t>& Gather(
            TGraphAllocator& graph_allocator, ResidualDatum& residual, RankDatum& current_ranks, UnusedData&... data)
        {
            graph_allocator.GatherDatum(current_ranks);
            return current_ranks.GetHostData();
        }

        template<
            typename ResidualDatum, typename RankDatum, typename...UnusedData>
        static std::vector<rank_t> Host(
            groute::graphs::host::CSRGraph& graph, ResidualDatum& residual, RankDatum& current_ranks, UnusedData&... data)
        {
            return PageRankHost(graph);
        }

        static int Output(const char *file, const std::vector<rank_t>& ranks)
        {
            return PageRankOutput(file, ranks);
        }

        static int CheckErrors(std::vector<rank_t>& ranks, std::vector<rank_t>& regression)
        {
            return PageRankCheckErrors(ranks, regression);
        }
    };

    using NodeResidualDatumType = groute::graphs::multi::NodeOutputGlobalDatum < rank_t > ;
    using NodeRankDatumType = groute::graphs::multi::NodeOutputLocalDatum < rank_t > ;

    template<bool IterationFusion = true, bool CTAScheduling = true>
    using FusedWorkerType = groute::FusedWorker <
        IterationFusion, local_work_t, remote_work_t, int, DWCallbacks, PageRankWork<CTAScheduling>,
        groute::graphs::dev::CSRGraphSeg, NodeResidualDatumType::DeviceObjectType, NodeRankDatumType::DeviceObjectType> ;
    
    template<bool CTAScheduling = true>
    using WorkerType = groute::Worker <
        local_work_t, remote_work_t, DWCallbacks, PageRankWork<CTAScheduling>,
        groute::graphs::dev::CSRGraphSeg, NodeResidualDatumType::DeviceObjectType, NodeRankDatumType::DeviceObjectType> ;

    template<typename TWorker>
    using RunnerType = utils::traversal::Runner <
        Algo, TWorker, DWCallbacks, local_work_t, remote_work_t,
        NodeResidualDatumType, NodeRankDatumType > ;
}

template<typename TWorker>
bool TestPageRankAsyncMultiTemplate(int ngpus)
{
    pr::RunnerType<TWorker> runner;

    pr::NodeResidualDatumType residual;
    pr::NodeRankDatumType ranks;

    return runner(ngpus, 0, residual, ranks);
}

bool TestPageRankAsyncMultiOptimized(int ngpus)
{
    return FLAGS_cta_np
        ? FLAGS_iteration_fusion
            ? TestPageRankAsyncMultiTemplate< pr::FusedWorkerType< true, true   >>(ngpus)
            : TestPageRankAsyncMultiTemplate< pr::FusedWorkerType< false, true  >>(ngpus)
        : FLAGS_iteration_fusion                               
            ? TestPageRankAsyncMultiTemplate< pr::FusedWorkerType< true, false  >>(ngpus)
            : TestPageRankAsyncMultiTemplate< pr::FusedWorkerType< false, false >>(ngpus);
}

bool TestPageRankAsyncMulti(int ngpus)
{
    return FLAGS_cta_np
        ? TestPageRankAsyncMultiTemplate< pr::WorkerType< true  >>(ngpus)
        : TestPageRankAsyncMultiTemplate< pr::WorkerType< false >>(ngpus);
}

bool TestPageRankSingle()
{
    return TestPageRankAsyncMultiOptimized(1);
}
