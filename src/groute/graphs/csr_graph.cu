// Groute: An Asynchronous Multi-GPU Programming Framework
// http://www.github.com/groute/groute
// Copyright (c) 2017, A. Barak
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice,
//   this list of conditions and the following disclaimer.
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
// * Neither the names of the copyright holders nor the names of its 
//   contributors may be used to endorse or promote products derived from this
//   software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
#ifdef HAVE_METIS
#include <metis.h>
#endif
#include <nvgraph.h>

#include <unordered_set>
#include <groute/graphs/csr_graph.h>
#include <cmath>

namespace groute {
namespace graphs {

    namespace multi
    {
        MetisPartitioner::MetisPartitioner(host::CSRGraph& origin_graph, int nsegs) : 
            m_origin_graph(origin_graph), 
            m_partitioned_graph(origin_graph.nnodes, origin_graph.nedges), 
            m_reverse_lookup(origin_graph.nnodes), m_seg_offsets(nsegs + 1),
            m_nsegs(nsegs)
        {
#ifndef HAVE_METIS
            printf("\nWARNING: Binary not built with METIS support. Exiting.\n");
            exit(100);
#else
            printf("\nStarting METIS partitioning\n");

            idx_t nnodes = m_origin_graph.nnodes;
            idx_t nedges = m_origin_graph.nedges;

            idx_t ncons = 1;
            idx_t nparts = m_nsegs;

            idx_t edgecut;
            std::vector<idx_t> partition_table(nnodes);

            // Convert to 64-bit for metis, idx_t is defined in metis.h
            std::vector<idx_t> row_start (nnodes+1), edge_dst (nedges), edge_weights;
            for (uint32_t i = 0; i < nnodes + 1; ++i)
                row_start[i] = static_cast<idx_t>(m_origin_graph.row_start[i]);
            for (uint32_t i = 0; i < nedges; ++i)
                edge_dst[i] = static_cast<idx_t>(m_origin_graph.edge_dst[i]);
            if(m_origin_graph.edge_weights)
            {
                edge_weights.resize(nedges);
                for (uint32_t i = 0; i < nedges; ++i)
                    edge_weights[i] = static_cast<idx_t>(m_origin_graph.edge_weights[i]);
            }
            printf("Converted graph to %d-bit, calling METIS\n", (int)IDXTYPEWIDTH);
            
            int result = METIS_PartGraphKway(
                &nnodes,                      // 
                &ncons,                       //
                row_start.data(),     //
                edge_dst.data(),      //
                NULL,                         // vwgt
                NULL,                         // vsize
                m_origin_graph.edge_weights ? edge_weights.data() : nullptr,  // adjwgt
                &nparts,                      // nparts
                NULL,                         // tpwgts
                NULL,                         // ubvec
                NULL,                         // options
                &edgecut,                     // objval
                &partition_table[0]);         // part [out]

            if (result != METIS_OK) {
                printf(
                    "METIS partitioning failed (%s error), Exiting.\n", 
                    result == METIS_ERROR_INPUT ? "input" : result == METIS_ERROR_MEMORY ? "memory" : "general");
                exit(0);
            }

            printf("Building partitioned graph and lookup tables\n");

            struct node_partition {
                    index_t node;
                    index_t partition;

                    node_partition(index_t node, index_t partition) : node(node), partition(partition) {}
                    node_partition() : node(-1), partition(-1) {}

                    inline bool operator< (const node_partition& rhs) const {
                        return partition < rhs.partition;
                    }
            };

            std::vector<node_partition> node_partitions(nnodes);

            for (index_t node = 0; node < nnodes; ++node)
            {
                node_partitions[node] = node_partition(node, partition_table[node]);
            }

            std::stable_sort(node_partitions.begin(), node_partitions.end());

            if (m_origin_graph.edge_weights != nullptr)
            {
                m_partitioned_graph.AllocWeights();
            }

            int current_seg = -1;

            for (index_t new_nidx = 0, edge_pos = 0; new_nidx < nnodes; ++new_nidx)
            {
                int seg = node_partitions[new_nidx].partition;
                while (seg > current_seg) // if this is true we have crossed the border to the next seg (looping with while just in case)
                {
                    m_seg_offsets[++current_seg] = new_nidx;
                }

                index_t origin_nidx = node_partitions[new_nidx].node; 
                m_reverse_lookup[origin_nidx] = new_nidx;

                index_t edge_start = m_origin_graph.row_start[origin_nidx];
                index_t edge_end = m_origin_graph.row_start[origin_nidx+1];

                m_partitioned_graph.row_start[new_nidx] = edge_pos;

                std::copy(m_origin_graph.edge_dst + edge_start, m_origin_graph.edge_dst + edge_end, m_partitioned_graph.edge_dst + edge_pos);

                if (m_origin_graph.edge_weights != nullptr) // copy weights
                    std::copy(m_origin_graph.edge_weights + edge_start, m_origin_graph.edge_weights + edge_end, m_partitioned_graph.edge_weights + edge_pos);

                edge_pos += (edge_end - edge_start);
            }
            
            while (m_nsegs > current_seg) m_seg_offsets[++current_seg] = nnodes;

            m_partitioned_graph.row_start[nnodes] = nedges;
            
            // Map the original destinations, copied from the origin graph to the new index space
            for (index_t edge = 0; edge < nedges; ++edge)
            {
                index_t origin_dest = m_partitioned_graph.edge_dst[edge];
                m_partitioned_graph.edge_dst[edge] = m_reverse_lookup[origin_dest];
            }

            printf("METIS partitioning done\n");
#endif
        }

        void MetisPartitioner::GetSegIndices(
            int seg_idx,
            index_t& seg_snode, index_t& seg_nnodes,
            index_t& seg_sedge, index_t& seg_nedges) const
        {
            index_t seg_enode, seg_eedge;

            seg_snode = m_seg_offsets[seg_idx];
            seg_enode = m_seg_offsets[seg_idx + 1];
            seg_nnodes = seg_enode - seg_snode;                

            seg_sedge = m_partitioned_graph.row_start[seg_snode];                            // start edge
            seg_eedge = m_partitioned_graph.row_start[seg_enode];                            // end edge
            seg_nedges = seg_eedge - seg_sedge;  
        }
        
        std::function<index_t(index_t)> MetisPartitioner::GetReverseLookupFunc()
        {
            return [this](index_t idx) { return this->m_reverse_lookup[idx]; };
        }

        std::vector<index_t> GetUniqueHalos(
            const index_t* edge_dst,
            index_t seg_snode, index_t seg_nnodes,
            index_t seg_sedge, index_t seg_nedges, int& halos_counter)
        {
            std::unordered_set<index_t> halos_set;
            halos_counter = 0;

            for (int i = 0; i < seg_nedges; ++i)
            {
                index_t dest = edge_dst[seg_sedge + i];
                if (dest < seg_snode || dest >= (seg_snode + seg_nnodes)) // an halo
                {
                    ++halos_counter;
                    halos_set.insert(dest);
                }
            }

            std::vector<index_t> halos_vec(halos_set.size());
            std::copy(halos_set.begin(), halos_set.end(), halos_vec.begin());

            return halos_vec;
        }

        // FQ
        NaivePartitioner::NaivePartitioner(host::CSRGraph& origin_graph, int nsegs) : 
            m_origin_graph(origin_graph), 
            m_partitioned_graph(origin_graph.nnodes, origin_graph.nedges), 
            m_reverse_lookup(origin_graph.nnodes), m_seg_offsets(nsegs + 1),
            m_nsegs(nsegs)
        {
#ifndef HAVE_METIS
            printf("\nWARNING: Binary not built with METIS support. Exiting.\n");
            exit(100);
#else
            printf("\nStarting naive partitioning\n");

            idx_t nnodes = m_origin_graph.nnodes;
            idx_t nedges = m_origin_graph.nedges;

            idx_t ncons = 1;
            idx_t nparts = m_nsegs;

            idx_t edgecut;
            std::vector<idx_t> partition_table(nnodes);

            // Convert to 64-bit for metis
            std::vector<idx_t> row_start (nnodes+1), edge_dst (nedges), edge_weights;
            for (uint32_t i = 0; i < nnodes + 1; ++i)
                row_start[i] = static_cast<idx_t>(m_origin_graph.row_start[i]);
            for (uint32_t i = 0; i < nedges; ++i)
                edge_dst[i] = static_cast<idx_t>(m_origin_graph.edge_dst[i]);
            if(m_origin_graph.edge_weights)
            {
                edge_weights.resize(nedges);
                for (uint32_t i = 0; i < nedges; ++i)
                    edge_weights[i] = static_cast<idx_t>(m_origin_graph.edge_weights[i]);
            }
            printf("Converted graph to %d-bit, doing naive partitioning\n", (int)IDXTYPEWIDTH);
            
            // int result = METIS_PartGraphKway(
            //     &nnodes,                      // 
            //     &ncons,                       //
            //     row_start.data(),     //
            //     edge_dst.data(),      //
            //     NULL,                         //
            //     NULL,                         //
            //     m_origin_graph.edge_weights ? edge_weights.data() : nullptr,  //
            //     &nparts,                      //
            //     NULL,                         //
            //     NULL,                         //
            //     NULL,                         //
            //     &edgecut,                     //
            //     &partition_table[0]);         //

            // FQ: we only need to modify partition_table
            idx_t nnodes_per_seg = nnodes / nparts;
            idx_t _t = 0;
            for (idx_t i = 0; i < nnodes; i++)
            {
                _t = i / nnodes_per_seg;
                if (_t >= nparts-1)
                    partition_table[i] = nparts-1;
                else
                    partition_table[i] = i / nnodes_per_seg;
                // printf("%d", partition_table[i]);
            }

            for (idx_t i = 0; i < 10; i++)
                printf("%d", partition_table[i]);

            printf("Building partitioned graph and lookup tables\n");

            // FQ: This struct store the relation between node ID and it partition ID
            struct node_partition {
                    index_t node;
                    index_t partition;

                    node_partition(index_t node, index_t partition) : node(node), partition(partition) {}
                    node_partition() : node(-1), partition(-1) {}

                    inline bool operator< (const node_partition& rhs) const {
                        return partition < rhs.partition;
                    }
            };

            std::vector<node_partition> node_partitions(nnodes);    // FQ: allocate 1 such struct for every node

            // FQ: init the data
            for (index_t node = 0; node < nnodes; ++node)
            {
                node_partitions[node] = node_partition(node, partition_table[node]);
            }

            // FQ: sort, put nodes belong to one partition together
            std::stable_sort(node_partitions.begin(), node_partitions.end());

            if (m_origin_graph.edge_weights != nullptr)
            {
                m_partitioned_graph.AllocWeights();
            }

            int current_seg = -1;

            // FQ: reorganize partitioned graph in CSR
            for (index_t new_nidx = 0, edge_pos = 0; new_nidx < nnodes; ++new_nidx)
            {
                int seg = node_partitions[new_nidx].partition;
                while (seg > current_seg) // if this is true we have crossed the border to the next seg (looping with while just in case)
                {
                    m_seg_offsets[++current_seg] = new_nidx;    // FQ: record the boundary node, the first node in seg
                }

                // FQ: construct a lookup table between old and new node ID, because we sort the node_partitions
                index_t origin_nidx = node_partitions[new_nidx].node; 
                m_reverse_lookup[origin_nidx] = new_nidx;       

                index_t edge_start = m_origin_graph.row_start[origin_nidx];
                index_t edge_end = m_origin_graph.row_start[origin_nidx+1];

                m_partitioned_graph.row_start[new_nidx] = edge_pos;

                std::copy(m_origin_graph.edge_dst + edge_start, m_origin_graph.edge_dst + edge_end, m_partitioned_graph.edge_dst + edge_pos);

                if (m_origin_graph.edge_weights != nullptr) // copy weights
                    std::copy(m_origin_graph.edge_weights + edge_start, m_origin_graph.edge_weights + edge_end, m_partitioned_graph.edge_weights + edge_pos);

                edge_pos += (edge_end - edge_start);
            }
            
            while (m_nsegs > current_seg) m_seg_offsets[++current_seg] = nnodes;

            m_partitioned_graph.row_start[nnodes] = nedges;
            
            // Map the original destinations, copied from the origin graph to the new index space
            for (index_t edge = 0; edge < nedges; ++edge)
            {
                index_t origin_dest = m_partitioned_graph.edge_dst[edge];
                m_partitioned_graph.edge_dst[edge] = m_reverse_lookup[origin_dest];
            }

            printf("Naive partitioning done\n");
#endif
        }

        void NaivePartitioner::GetSegIndices(
            int seg_idx,
            index_t& seg_snode, index_t& seg_nnodes,
            index_t& seg_sedge, index_t& seg_nedges) const
        {
            index_t seg_enode, seg_eedge;

            seg_snode = m_seg_offsets[seg_idx];
            seg_enode = m_seg_offsets[seg_idx + 1];
            seg_nnodes = seg_enode - seg_snode;                

            seg_sedge = m_partitioned_graph.row_start[seg_snode];                            // start edge
            seg_eedge = m_partitioned_graph.row_start[seg_enode];                            // end edge
            seg_nedges = seg_eedge - seg_sedge;  
        }
        
        std::function<index_t(index_t)> NaivePartitioner::GetReverseLookupFunc()
        {
            return [this](index_t idx) { return this->m_reverse_lookup[idx]; };
        }

        MetisPartitionerDegreeW::MetisPartitionerDegreeW(host::CSRGraph& origin_graph, int nsegs) : 
            m_origin_graph(origin_graph), 
            m_partitioned_graph(origin_graph.nnodes, origin_graph.nedges), 
            m_reverse_lookup(origin_graph.nnodes), m_seg_offsets(nsegs + 1),
            m_nsegs(nsegs)
        {
#ifndef HAVE_METIS
            printf("\nWARNING: Binary not built with METIS support. Exiting.\n");
            exit(100);
#else
            printf("\nStarting METIS partitioning with vertex weights (degree)\n");

            idx_t nnodes = m_origin_graph.nnodes;
            idx_t nedges = m_origin_graph.nedges;

            idx_t ncons = 1;
            idx_t nparts = m_nsegs;

            printf("0..");
            fflush(stdout);
            idx_t edgecut;
            std::vector<idx_t> partition_table(nnodes);

            printf("1..");
            fflush(stdout);

            // Convert to 64-bit for metis
            std::vector<idx_t> row_start (nnodes+1), edge_dst (nedges), edge_weights;
            for (uint32_t i = 0; i < nnodes + 1; ++i)
                row_start[i] = static_cast<idx_t>(m_origin_graph.row_start[i]);
            printf("2..");fflush(stdout);
            for (uint32_t i = 0; i < nedges; ++i)
                edge_dst[i] = static_cast<idx_t>(m_origin_graph.edge_dst[i]);
            printf("3..");fflush(stdout);
            if(m_origin_graph.edge_weights)
            {
                edge_weights.resize(nedges);
                for (uint32_t i = 0; i < nedges; ++i)
                    edge_weights[i] = static_cast<idx_t>(m_origin_graph.edge_weights[i]);
            }
            printf("Converted graph to %d-bit, computing degrees\n", (int)IDXTYPEWIDTH);

            idx_t *vdegrees = (idx_t*) malloc(sizeof(idx_t) * nnodes);
            for (idx_t i = 0; i < nnodes; i++)
            {
                vdegrees[i] = m_origin_graph.row_start[i+1] - m_origin_graph.row_start[i];
            }

            printf("Degree computed, calling METIS\n"); 
            
            int result = METIS_PartGraphKway(
                &nnodes,                      // 
                &ncons,                       //
                row_start.data(),     //
                edge_dst.data(),      //
                vdegrees,                         // vwgt
                NULL,                         // vsize
                m_origin_graph.edge_weights ? edge_weights.data() : nullptr,  // adjwgt
                &nparts,                      // nparts
                NULL,                         // tpwgts
                NULL,                         // ubvec
                NULL,                         // options
                &edgecut,                     // objval
                &partition_table[0]);         // part [out]

            if (result != METIS_OK) {
                printf(
                    "METIS partitioning failed (%s error), Exiting.\n", 
                    result == METIS_ERROR_INPUT ? "input" : result == METIS_ERROR_MEMORY ? "memory" : "general");
                exit(0);
            }
            // free(vdegrees);

            printf("Building partitioned graph and lookup tables\n");

            struct node_partition {
                    index_t node;
                    index_t partition;

                    node_partition(index_t node, index_t partition) : node(node), partition(partition) {}
                    node_partition() : node(-1), partition(-1) {}

                    inline bool operator< (const node_partition& rhs) const {
                        return partition < rhs.partition;
                    }
            };

            std::vector<node_partition> node_partitions(nnodes);

            for (index_t node = 0; node < nnodes; ++node)
            {
                node_partitions[node] = node_partition(node, partition_table[node]);
            }

            std::stable_sort(node_partitions.begin(), node_partitions.end());

            if (m_origin_graph.edge_weights != nullptr)
            {
                m_partitioned_graph.AllocWeights();
            }

            int current_seg = -1;

            for (index_t new_nidx = 0, edge_pos = 0; new_nidx < nnodes; ++new_nidx)
            {
                int seg = node_partitions[new_nidx].partition;
                while (seg > current_seg) // if this is true we have crossed the border to the next seg (looping with while just in case)
                {
                    m_seg_offsets[++current_seg] = new_nidx;
                }

                index_t origin_nidx = node_partitions[new_nidx].node; 
                m_reverse_lookup[origin_nidx] = new_nidx;

                index_t edge_start = m_origin_graph.row_start[origin_nidx];
                index_t edge_end = m_origin_graph.row_start[origin_nidx+1];

                m_partitioned_graph.row_start[new_nidx] = edge_pos;

                std::copy(m_origin_graph.edge_dst + edge_start, m_origin_graph.edge_dst + edge_end, m_partitioned_graph.edge_dst + edge_pos);

                if (m_origin_graph.edge_weights != nullptr) // copy weights
                    std::copy(m_origin_graph.edge_weights + edge_start, m_origin_graph.edge_weights + edge_end, m_partitioned_graph.edge_weights + edge_pos);

                edge_pos += (edge_end - edge_start);
            }
            
            while (m_nsegs > current_seg) m_seg_offsets[++current_seg] = nnodes;

            m_partitioned_graph.row_start[nnodes] = nedges;
            
            // Map the original destinations, copied from the origin graph to the new index space
            for (index_t edge = 0; edge < nedges; ++edge)
            {
                index_t origin_dest = m_partitioned_graph.edge_dst[edge];
                m_partitioned_graph.edge_dst[edge] = m_reverse_lookup[origin_dest];
            }

            printf("METIS partitioning done\n");
#endif
        }

        void MetisPartitionerDegreeW::GetSegIndices(
            int seg_idx,
            index_t& seg_snode, index_t& seg_nnodes,
            index_t& seg_sedge, index_t& seg_nedges) const
        {
            index_t seg_enode, seg_eedge;

            seg_snode = m_seg_offsets[seg_idx];
            seg_enode = m_seg_offsets[seg_idx + 1];
            seg_nnodes = seg_enode - seg_snode;                

            seg_sedge = m_partitioned_graph.row_start[seg_snode];                            // start edge
            seg_eedge = m_partitioned_graph.row_start[seg_enode];                            // end edge
            seg_nedges = seg_eedge - seg_sedge;  
        }
        
        std::function<index_t(index_t)> MetisPartitionerDegreeW::GetReverseLookupFunc()
        {
            return [this](index_t idx) { return this->m_reverse_lookup[idx]; };
        }

/* --------------------------------- NVGraph -------------------------------- */
        void check_status(nvgraphStatus_t status)
        {
            // printf("check status\n");
            if (status != NVGRAPH_STATUS_SUCCESS)
            {
                printf("ERROR : %s\n", nvgraphStatusGetString(status));
                exit(0);
            }
        }

        NVGraphPartitioner::NVGraphPartitioner(host::CSRGraph& origin_graph, int nsegs) : 
            m_origin_graph(origin_graph), 
            m_partitioned_graph(origin_graph.nnodes, origin_graph.nedges), 
            m_reverse_lookup(origin_graph.nnodes), m_seg_offsets(nsegs + 1),
            m_nsegs(nsegs)
        {
            printf("\nStarting NVGraph partitioning with vertex weights (degree)\n");

            int nnodes = m_origin_graph.nnodes;
            int nedges = m_origin_graph.nedges;

            int ncons = 1;
            int nparts = m_nsegs;

            printf("0..");
            fflush(stdout);
            int edgecut;
            std::vector<int> partition_table(nnodes);

            printf("1..");
            fflush(stdout);

            // Convert to int for nvgraph
            std::vector<int> row_start (nnodes+1), edge_dst (nedges), edge_weights;
            for (uint32_t i = 0; i < nnodes + 1; ++i)
                row_start[i] = static_cast<int>(m_origin_graph.row_start[i]);
            printf("2..");fflush(stdout);
            for (uint32_t i = 0; i < nedges; ++i)
                edge_dst[i] = static_cast<int>(m_origin_graph.edge_dst[i]);
            printf("3..");fflush(stdout);
            if(m_origin_graph.edge_weights)
            {
                edge_weights.resize(nedges);
                for (uint32_t i = 0; i < nedges; ++i)
                    edge_weights[i] = static_cast<int>(m_origin_graph.edge_weights[i]);
            }
            printf("Converted graph to %d-bit, calling nvgraph...\n", (int)IDXTYPEWIDTH);

            // idx_t *vdegrees = (idx_t*) malloc(sizeof(idx_t) * nnodes);
            // for (idx_t i = 0; i < nnodes; i++)
            // {
            //     vdegrees[i] = m_origin_graph.row_start[i+1] - m_origin_graph.row_start[i];
            // }

            // printf("Degree computed, calling METIS\n"); 

            nvgraphHandle_t handle;
            nvgraphGraphDescr_t graph;
            nvgraphCSRTopology32I_st CSRType;
            CSRType.nvertices = nnodes;
            CSRType.nedges = ncons;
            CSRType.source_offsets = row_start.data();
            CSRType.destination_indices = edge_dst.data();

            SpectralClusteringParameter param;
            param.n_clusters = nparts;
            param.n_eig_vects = nparts;
            param.algorithm = NVGRAPH_BALANCED_CUT_LANCZOS;
            param.evs_tolerance = 0.0f; // default
            param.evs_max_iter = 0; // default
            param.kmean_tolerance = 0.0f;   // default
            param.kmean_max_iter = 0;   // default
            
            hipDataType edge_t = HIP_R_32F;

            // allocate
            float *edgewgt, *eigvals, *eigvec;
            edgewgt = (float*) malloc(sizeof(float) * ncons);
            eigvals = (float*) malloc(sizeof(float)*nparts);
            eigvec = (float*) malloc(sizeof(float)*nparts*nnodes);
            for (uint32_t i = 0; i < ncons; i++)
            {
                edgewgt[i] = 1;
            }
            printf("prepared...");fflush(stdout);

            check_status(nvgraphCreate(&handle));
            check_status(nvgraphCreateGraphDescr(handle, &graph));
            check_status(nvgraphSetGraphStructure(handle, graph, (void*)&CSRType, NVGRAPH_CSR_32));
            check_status(nvgraphAllocateEdgeData(handle, graph, 1, &edge_t));
            check_status(nvgraphSetEdgeData(handle, graph, (void*)edgewgt, 0));
            printf("running...");fflush(stdout);
            check_status(nvgraphSpectralClustering(handle, graph, 0, &param, &partition_table[0], eigvals, eigvec));

            check_status(nvgraphDestroyGraphDescr(handle, graph));
            check_status(nvgraphDestroy(handle));
            free(edgewgt);
            free(eigvals);
            free(eigvec);
            
            // int result = METIS_PartGraphKway(
            //     &nnodes,                      // 
            //     &ncons,                       //
            //     row_start.data(),     //
            //     edge_dst.data(),      //
            //     NULL,                         // vwgt
            //     NULL,                         // vsize
            //     m_origin_graph.edge_weights ? edge_weights.data() : nullptr,  // adjwgt
            //     &nparts,                      // nparts
            //     NULL,                         // tpwgts
            //     NULL,                         // ubvec
            //     NULL,                         // options
            //     &edgecut,                     // objval
            //     &partition_table[0]);         // part [out]

            // if (result != METIS_OK) {
            //     printf(
            //         "METIS partitioning failed (%s error), Exiting.\n", 
            //         result == METIS_ERROR_INPUT ? "input" : result == METIS_ERROR_MEMORY ? "memory" : "general");
            //     exit(0);
            // }
            // free(vdegrees);

            printf("Building partitioned graph and lookup tables\n");

            struct node_partition {
                    index_t node;
                    index_t partition;

                    node_partition(index_t node, index_t partition) : node(node), partition(partition) {}
                    node_partition() : node(-1), partition(-1) {}

                    inline bool operator< (const node_partition& rhs) const {
                        return partition < rhs.partition;
                    }
            };

            std::vector<node_partition> node_partitions(nnodes);

            for (index_t node = 0; node < nnodes; ++node)
            {
                node_partitions[node] = node_partition(node, partition_table[node]);
            }

            std::stable_sort(node_partitions.begin(), node_partitions.end());

            if (m_origin_graph.edge_weights != nullptr)
            {
                m_partitioned_graph.AllocWeights();
            }

            int current_seg = -1;

            for (index_t new_nidx = 0, edge_pos = 0; new_nidx < nnodes; ++new_nidx)
            {
                int seg = node_partitions[new_nidx].partition;
                while (seg > current_seg) // if this is true we have crossed the border to the next seg (looping with while just in case)
                {
                    m_seg_offsets[++current_seg] = new_nidx;
                }

                index_t origin_nidx = node_partitions[new_nidx].node; 
                m_reverse_lookup[origin_nidx] = new_nidx;

                index_t edge_start = m_origin_graph.row_start[origin_nidx];
                index_t edge_end = m_origin_graph.row_start[origin_nidx+1];

                m_partitioned_graph.row_start[new_nidx] = edge_pos;

                std::copy(m_origin_graph.edge_dst + edge_start, m_origin_graph.edge_dst + edge_end, m_partitioned_graph.edge_dst + edge_pos);

                if (m_origin_graph.edge_weights != nullptr) // copy weights
                    std::copy(m_origin_graph.edge_weights + edge_start, m_origin_graph.edge_weights + edge_end, m_partitioned_graph.edge_weights + edge_pos);

                edge_pos += (edge_end - edge_start);
            }
            
            while (m_nsegs > current_seg) m_seg_offsets[++current_seg] = nnodes;

            m_partitioned_graph.row_start[nnodes] = nedges;
            
            // Map the original destinations, copied from the origin graph to the new index space
            for (index_t edge = 0; edge < nedges; ++edge)
            {
                index_t origin_dest = m_partitioned_graph.edge_dst[edge];
                m_partitioned_graph.edge_dst[edge] = m_reverse_lookup[origin_dest];
            }

            printf("NVGraph partitioning done\n");
        }

        void NVGraphPartitioner::GetSegIndices(
            int seg_idx,
            index_t& seg_snode, index_t& seg_nnodes,
            index_t& seg_sedge, index_t& seg_nedges) const
        {
            index_t seg_enode, seg_eedge;

            seg_snode = m_seg_offsets[seg_idx];
            seg_enode = m_seg_offsets[seg_idx + 1];
            seg_nnodes = seg_enode - seg_snode;                

            seg_sedge = m_partitioned_graph.row_start[seg_snode];                            // start edge
            seg_eedge = m_partitioned_graph.row_start[seg_enode];                            // end edge
            seg_nedges = seg_eedge - seg_sedge;  
        }
        
        std::function<index_t(index_t)> NVGraphPartitioner::GetReverseLookupFunc()
        {
            return [this](index_t idx) { return this->m_reverse_lookup[idx]; };
        }
    }   // namespace multi
}
}
